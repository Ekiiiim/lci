#include "hip/hip_runtime.h"
// Copyright (c) 2025 The LCI Project Authors
// SPDX-License-Identifier: MIT

#include "lci.hpp"
#include "cuda_util.hpp"

__global__ void init_buffer(float* d_buf, float value, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_buf[idx] = value;
    }
}

__global__ void verify_buffer(float* d_buf, float expected, int N, int* d_flag) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    if (d_buf[idx] != expected) {
      *d_flag = 1;
    }
  }
}

void verify_buffer(float* d_buf, float expected, int N) {
  int* d_flag = nullptr;
  CUDA_CHECK(hipMalloc(&d_flag, sizeof(int)));
  CUDA_CHECK(hipMemset(d_flag, 0, sizeof(int)));
  verify_buffer<<<1, N>>>(d_buf, expected, N, d_flag);
  int h_flag = 0;
  CUDA_CHECK(hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_flag));
  if (h_flag == 1) {
    printf("Error: buffer verification failed\n");
    exit(1);
  }
}

int main(int argc, char** argv)
{
  const int msg_size = 65536;

  lci::g_runtime_init();
  int rank_me = lci::get_rank_me();
  int rank_n = lci::get_rank_n();
  
  // allocate buffers
  float* src_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&src_buffer, msg_size));
  CUDA_CHECK(hipMemset(src_buffer, 0, msg_size));

  float* dst_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&dst_buffer, msg_size));
  CUDA_CHECK(hipMemset(dst_buffer, 0, msg_size));

  // initialize source buffer
  init_buffer<<<1, msg_size>>>(src_buffer, rank_me, msg_size);
  CUDA_CHECK(hipDeviceSynchronize());

  // ring, should also work for rank_n == 1
  int left = (rank_me - 1 + rank_n) % rank_n;
  int right = (rank_me + 1) % rank_n;
  lci::comp_t sync = lci::alloc_sync_x().threshold(2)();
  lci::post_send_x(left, src_buffer, msg_size, 0, sync).allow_retry(false)();
  lci::post_recv_x(right, dst_buffer, msg_size, 0, sync).allow_retry(false)();
  lci::sync_wait(sync, nullptr);
  verify_buffer(dst_buffer, right, msg_size);
  
  CUDA_CHECK(hipFree(src_buffer));
  CUDA_CHECK(hipFree(dst_buffer));

  lci::g_runtime_fina();
  return 0;
}