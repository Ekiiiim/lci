#include "hip/hip_runtime.h"
// Copyright (c) 2025 The LCI Project Authors
// SPDX-License-Identifier: MIT

#include <iostream>
#include <unistd.h>
#include <cassert>

#include "lci.hpp"
#include "cuda_util.hpp"

__global__ void init_buffer(float* d_buf, float value, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        d_buf[idx] = value;
    }
}

__global__ void verify_buffer(float* d_buf, float expected, int N, int* d_flag) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N) {
    if (d_buf[idx] != expected) {
      *d_flag = 1;
    }
  }
}

void verify_buffer(float* d_buf, float expected, int N) {
  int* d_flag = nullptr;
  CUDA_CHECK(hipMalloc(&d_flag, sizeof(int)));
  CUDA_CHECK(hipMemset(d_flag, 0, sizeof(int)));
  verify_buffer<<<1, N>>>(d_buf, expected, N, d_flag);
  int h_flag = 0;
  CUDA_CHECK(hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(d_flag));
  if (h_flag == 1) {
    printf("Error: buffer verification failed\n");
    exit(1);
  }
}

int main(int argc, char** argv)
{
  int msg_size = 65536;
  if (argc >= 2) {
    msg_size = atoi(argv[1]);
  }

  assert(msg_size % sizeof(float) == 0);
  int N = msg_size / sizeof(float);

  lci::g_runtime_init();
  int rank_me = lci::get_rank_me();
  int rank_n = lci::get_rank_n();
  
  // allocate buffers
  float* src_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&src_buffer, msg_size));
  CUDA_CHECK(hipMemset(src_buffer, -1, msg_size));

  float* dst_buffer = nullptr;
  CUDA_CHECK(hipMalloc(&dst_buffer, msg_size));
  CUDA_CHECK(hipMemset(dst_buffer, -1, msg_size));

  // initialize source buffer
  init_buffer<<<1, N>>>(src_buffer, rank_me, N);
  CUDA_CHECK(hipDeviceSynchronize());

  // ring, should also work for rank_n == 1
  int left = (rank_me - 1 + rank_n) % rank_n;
  int right = (rank_me + 1) % rank_n;
  lci::comp_t sync = lci::alloc_sync_x().threshold(2)();
  lci::post_send_x(left, src_buffer, msg_size, 0, sync).allow_retry(false).mr(lci::MR_DEVICE)();
  lci::post_recv_x(right, dst_buffer, msg_size, 0, sync).allow_retry(false).mr(lci::MR_DEVICE)();
  lci::sync_wait(sync, nullptr);
  verify_buffer(dst_buffer, right, N);

  if (rank_me == 0) {
    std::cout << "Test passed" << std::endl;
    std::cout << "message size: " << msg_size << std::endl;
    std::cout << "rank_n: " << rank_n << std::endl;
  }
  
  CUDA_CHECK(hipFree(src_buffer));
  CUDA_CHECK(hipFree(dst_buffer));

  lci::g_runtime_fina();
  return 0;
}