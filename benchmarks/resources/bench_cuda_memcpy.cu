#include <getopt.h>
#include <thread>
#include <chrono>
#include <hip/hip_runtime.h>

#include "lct.h"
#include "lci.hpp"

#include "util.hpp"

#define CUDA_CHECK(call)                                                \
  do {                                                                  \
    hipError_t err = call;                                             \
    if (hipSuccess != err) {                                           \
      fprintf(stderr, "Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
              hipGetErrorString(err));                                 \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  } while (0)

enum class memcpy_type_t {
  HOST_TO_HOST,
  HOST_TO_DEVICE,
  DEVICE_TO_HOST,
  DEVICE_TO_DEVICE,
};

struct config_t {
  int nthreads = 16;
  int niters = 1;
  size_t total_size = 1024 * 1024 * 1024; // Total size in bytes
  int block_size = 4096;
  memcpy_type_t memcpy_type = memcpy_type_t::HOST_TO_DEVICE;
} config;

LCT_tbarrier_t g_tbarrier;

char *src_buffer;
char *dst_buffer;

void worker(int id) {
  util::pin_thread_to_cpu(id);
  size_t total_accesses = config.total_size / config.block_size;
  size_t accesses_per_thread = total_accesses / config.nthreads;
  size_t start_access = id * accesses_per_thread;
  LCT_tbarrier_arrive_and_wait(g_tbarrier);
  auto start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < config.niters; i++) {
    for (int j = 0; j < accesses_per_thread; j++) {
      size_t access = start_access + j;
      size_t offset = access * config.block_size;
      switch (config.memcpy_type) {
        case memcpy_type_t::HOST_TO_HOST:
          memcpy(dst_buffer + offset, src_buffer + offset, config.block_size);
          break;
        case memcpy_type_t::HOST_TO_DEVICE:
          CUDA_CHECK(hipMemcpy(dst_buffer + offset, src_buffer + offset, config.block_size, hipMemcpyHostToDevice));
          break;
        case memcpy_type_t::DEVICE_TO_HOST:
          CUDA_CHECK(hipMemcpy(dst_buffer + offset, src_buffer + offset, config.block_size, hipMemcpyDeviceToHost));
          break;
        case memcpy_type_t::DEVICE_TO_DEVICE:
          CUDA_CHECK(hipMemcpy(dst_buffer + offset, src_buffer + offset, config.block_size, hipMemcpyDeviceToDevice));
      }
    }
  }
  LCT_tbarrier_arrive_and_wait(g_tbarrier);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  double elapsed_s = elapsed.count();
  if (id == 0) {
    printf("Elapsed time: %.2f s\n", elapsed_s);
    printf("Per-operation time: %.2f us\n",
           (elapsed_s * 1e6) / (config.niters * accesses_per_thread));
    double throughput_per_thread = (static_cast<double>(config.niters) * accesses_per_thread) / (elapsed_s * 1e6);
    printf("Throughput per thread: %.2f Mops/s\n",
           throughput_per_thread);
    printf("Throughput: %.2f Mops/s\n", throughput_per_thread * config.nthreads);
    printf("Bandwidth per thread: %.2f GB/s\n",
           (throughput_per_thread * config.block_size) / 1e3);
    printf("Bandwidth: %.2f GB/s\n",
           (throughput_per_thread * config.nthreads * config.block_size) / 1e3);
  }
}

int main(int argc, char** argv) {
  int total_size_mb = config.total_size / (1024 * 1024);
  LCT_args_parser_t argsParser = LCT_args_parser_alloc();
  LCT_args_parser_add(argsParser, "nthreads", required_argument,
    &config.nthreads);
  LCT_args_parser_add(argsParser, "niters", required_argument,
      &config.niters);
  LCT_args_parser_add(argsParser, "total-size", required_argument,
      &total_size_mb);
  LCT_args_parser_add(argsParser, "block-size", required_argument,
      &config.block_size);
  LCT_dict_str_int_t memcpy_type_dict[] = {
    {"h2h", static_cast<int>(memcpy_type_t::HOST_TO_HOST)},
    {"h2d", static_cast<int>(memcpy_type_t::HOST_TO_DEVICE)},
    {"d2h", static_cast<int>(memcpy_type_t::DEVICE_TO_HOST)},
    {"d2d", static_cast<int>(memcpy_type_t::DEVICE_TO_DEVICE)},
  };
  LCT_args_parser_add_dict(argsParser, "type", required_argument,
      reinterpret_cast<int*>(&config.memcpy_type), memcpy_type_dict, 4);
  LCT_args_parser_parse(argsParser, argc, argv);
  LCT_args_parser_print(argsParser, true);
  LCT_args_parser_free(argsParser);

  config.total_size = static_cast<size_t>(total_size_mb) * 1024 * 1024; // Convert to bytes
  g_tbarrier = LCT_tbarrier_alloc(config.nthreads);
  switch (config.memcpy_type) {
    case memcpy_type_t::HOST_TO_HOST:
      src_buffer = static_cast<char*>(aligned_alloc(4096, config.total_size));
      dst_buffer = static_cast<char*>(aligned_alloc(4096, config.total_size));
      break;
    case memcpy_type_t::HOST_TO_DEVICE:
      src_buffer = static_cast<char*>(aligned_alloc(4096, config.total_size));
      CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&dst_buffer), config.total_size));
      break;
    case memcpy_type_t::DEVICE_TO_HOST:
      CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&src_buffer), config.total_size));
      dst_buffer = static_cast<char*>(aligned_alloc(4096, config.total_size));
      break;
    case memcpy_type_t::DEVICE_TO_DEVICE:
      CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&src_buffer), config.total_size));
      CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&dst_buffer), config.total_size));
      break;
  }

  lci::g_runtime_init();

  std::vector<std::thread> threads;
  for (int i = 0; i < config.nthreads; i++) {
    std::thread t(worker, i);
    threads.push_back(std::move(t));
  }
  for (auto& t : threads) {
    t.join();
  }

  lci::g_runtime_fina();

  switch (config.memcpy_type) {
    case memcpy_type_t::HOST_TO_HOST:
      free(src_buffer);
      free(dst_buffer);
      break;
    case memcpy_type_t::HOST_TO_DEVICE:
      free(src_buffer);
      CUDA_CHECK(hipFree(dst_buffer));
      break;
    case memcpy_type_t::DEVICE_TO_HOST:
      CUDA_CHECK(hipFree(src_buffer));
      free(dst_buffer);
      break;
    case memcpy_type_t::DEVICE_TO_DEVICE:
      CUDA_CHECK(hipFree(src_buffer));
      CUDA_CHECK(hipFree(dst_buffer));
      break;
  }

  LCT_tbarrier_free(&g_tbarrier);
  return 0;
}